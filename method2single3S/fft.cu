#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hipfft/hipfft.h>
#include<hipblas.h>
#include<iostream>

#define BATCH 1
__global__ void real2complex(hipfftDoubleReal *in, hipfftDoubleComplex *out) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	out[i] = make_hipDoubleComplex(in[i], 0);
}

__global__ void real2complex_(const hipfftDoubleReal* __restrict__ in, hipfftDoubleComplex* __restrict__ out, int NN) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;

#pragma unroll
	for (; i < NN; i += blockDim.x*gridDim.x) {
		out[i] = make_hipDoubleComplex(in[i], 0);
	}

}

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}

void cuFFTR2C(hipStream_t stream, hipfftHandle plan, hipfftDoubleReal *indata, hipDoubleComplex *outdata, int Nslice, int My, int Mx) {

	int nElem = Nslice * My*Mx;
	//dim3 block(64);
	real2complex << < (nElem / 128), 128, 0, stream >> > (indata, outdata);



	//hipfftResult status;
	//status = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)outdata, (hipfftDoubleComplex*)outdata, HIPFFT_FORWARD);
	//std::cout << status << std::endl;

	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}



}

void cuFFTR2C_1(hipStream_t stream, hipfftHandle plan, hipfftDoubleReal *indata, hipDoubleComplex *outdata_, hipDoubleComplex *outdata, int Nslice, int My, int Mx) {

	int nElem = Nslice * My*Mx;
	//dim3 block(64);
	real2complex << < (nElem / 128), 128, 0, stream >> > (indata, outdata_);



	//hipfftResult status;
	//status = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)outdata, (hipfftDoubleComplex*)outdata, HIPFFT_FORWARD);
	//std::cout << status << std::endl;

	if (hipfftExecZ2Z(plan, outdata_, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}



}


void cuFFTR2C_(hipStream_t stream, hipfftHandle plan, hipfftDoubleReal *indata, hipDoubleComplex *outdata, int Nslice, int My, int Mx) {

	int nElem = Nslice * My*Mx;
	//dim3 block(64);
	real2complex_ << < (nElem / 1024), 128, 0, stream >> > (indata, outdata, nElem);



	//hipfftResult status;
	//status = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)outdata, (hipfftDoubleComplex*)outdata, HIPFFT_FORWARD);
	//std::cout << status << std::endl;

	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}



}

void cuFFTC2C(hipStream_t stream, hipfftHandle plan, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {


	//hipfftResult status;
	//status = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)outdata, (hipfftDoubleComplex*)outdata, HIPFFT_FORWARD);
	//std::cout << status << std::endl;

	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}

}

void cuIFFTC2C(hipStream_t stream, hipfftHandle plan, hipblasHandle_t handle, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {   // because C-> C we only need 1 variable: outdata. 


																																		  //hipblasHandle_t handle;
																																		  //hipblasStatus_t status;
	double alpha = double(1) / (Nslice*My*Mx);



	//status = hipblasCreate(&handle);
	//hipfftResult status;
	//status = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)outdata, (hipfftDoubleComplex*)outdata, HIPFFT_BACKWARD);
	//std::cout << status << std::endl;


	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z inverse failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}



	//std::cout <<"FFT Error" <<_cudaGetErrorEnum(hipblasZdscal(handle, Nslice*My*Mx, &alpha, outdata, 1)) << std::endl;
	hipblasZdscal(handle, Nslice*My*Mx, &alpha, outdata, 1);
}