#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "fft.h"
#include <stdlib.h>
#include <iostream>



__global__ void subsref(hipfftDoubleComplex* indata, hipfftDoubleComplex *outdata, double *S, long nnz) {
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<nnz)
		outdata[idx] = indata[(long)S[idx]];
}
__global__ void subsasgn(hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {
	//*data must be memset
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<nnz)
		outdata[(long)S[idx]] = indata[idx];

}
__global__ void piecewiseMatMul(hipfftDoubleReal *ref, hipfftDoubleComplex *mat) {
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	mat[idx].x = ref[idx] * mat[idx].x;
	mat[idx].y = ref[idx] * mat[idx].y;
}

__global__ void piecewiseMatMul_(const double* __restrict__ ref, double2* __restrict__ mat, int NN) {
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	//double tmpx, tmpy;
#pragma unroll
	for (; idx < NN; idx += blockDim.x*gridDim.x) {
		mat[idx].x = ref[idx] * mat[idx].x;
		mat[idx].y = ref[idx] * mat[idx].y;
		/*double tmpx = mat[idx].x;
		double tmpy = mat[idx].y;

		tmpx *= ref[idx];
		tmpy *= ref[idx];

		mat[idx].x = tmpx;
		mat[idx].y = tmpy;*/
	}


}

__global__ void subsref_(const hipfftDoubleComplex* __restrict__ indata, hipfftDoubleComplex* __restrict__ outdata, const double* __restrict__ S, long nnz) {
	long idx = blockIdx.x*blockDim.x + threadIdx.x;

#pragma unroll
	for (; idx < nnz; idx += blockDim.x*gridDim.x) {
		outdata[idx] = indata[(long)S[idx]];
	}

}
__global__ void subsasgn_(const hipfftDoubleComplex* __restrict__ indata, hipfftDoubleComplex* __restrict__ outdata, const double* __restrict__ S, long nnz) {
	//*data must be memset
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
#pragma unroll
	for (; idx < nnz; idx += blockDim.x*gridDim.x) {
		outdata[(long)S[idx]] = indata[idx];
	}


}

void AopReal(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleReal *indata, hipfftDoubleComplex *outdata_, hipfftDoubleComplex *outdata, double *S, long nnz) {



	cuFFTR2C_(stream, plan, indata, outdata_, Nslice, My, Mx);


	subsref_ << <((nnz + 1024 - 1) / 1024), 128, 0, stream >> > (outdata_, outdata, S, nnz);
	//hipDeviceSynchronize(); 
	//hipFree(outdata_); 


}

void AopCompl(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {

	//hipfftComplex *outdata_; 
	//hipMalloc(&outdata_, Nslice*My*Mx * sizeof(hipfftComplex));
	//AWARE:  INPUT OF AOPCOMPL IS CHANGED, SO DON'T USE INPUT AGAIN!!!! IF YOU WANT TO USE, COPY IT TO ANOTHER VARIABLE. 
	//hipfftComplex *outdata_; 
	//hipMalloc(&outdata_, Nslice*My*Mx * sizeof(hipfftComplex));

	cuFFTC2C(stream, plan, indata, Nslice, My, Mx);

	subsref_ << <((nnz + 1024 - 1) / 1024), 128, 0, stream >> > (indata, outdata, S, nnz);
	//hipDeviceSynchronize();


	//hipFree(outdata_); 

}

void Ahop(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {

	hipMemsetAsync(outdata, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex), stream);
	subsasgn_ << <((nnz + 1024 - 1) / 1024), 128, 0, stream >> > (indata, outdata, S, nnz);


	//hipDeviceSynchronize();

	cuIFFTC2C(stream, plan, handle, outdata, Nslice, My, Mx);



}

void Ahop_(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {

	hipMemsetAsync(outdata, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex), stream);
	subsasgn_ << <((nnz + 1024 - 1) / 1024), 128, 0, stream >> > (indata, outdata, S, nnz);


	//hipDeviceSynchronize();

	cuIFFTC2C(stream, plan, handle, outdata, Nslice, My, Mx);



}

void Bhop_gs(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleReal *ref, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata_, hipfftDoubleComplex *outdata, double *S, long nnz) {



	//hipMemsetAsync(outdata_, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex), stream);

	//hipMemset(outdata_, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex));
	Ahop(Nslice, My, Mx, plan, handle, stream, indata, outdata_, S, nnz);

	piecewiseMatMul_ << <(Nslice*My*Mx / 1024), 128, 0, stream >> >(ref, outdata_, Nslice*My*Mx);
	//hipDeviceSynchronize();
	AopCompl(Nslice, My, Mx, plan, handle, stream, outdata_, outdata, S, nnz);
	//hipFree(outdata_);

}
void Iop_gs(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleReal *ref, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {

	//hipMemset(outdata, 0, Nslice*My*Mx * sizeof(hipfftComplex));
	//hipMemsetAsync(outdata, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex), stream);

	Ahop(Nslice, My, Mx, plan, handle, stream, indata, outdata, S, nnz);
	piecewiseMatMul_ << <(Nslice*My*Mx / 1024), 128, 0, stream >> >(ref, outdata, Nslice*My*Mx);

}